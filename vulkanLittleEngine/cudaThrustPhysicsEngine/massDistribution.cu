#define GLM_FORCE_RADIANS
#define GLM_FORCE_DEPTH_ZERO_TO_ONE
#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <vector>
#include <array>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <algorithm>
#include <stdexcept>
#include "physicsEngine.cuh"
#include "exceptionHandling.h"

/*
------------------- RIGID BODY PHYSICS -------------------
			DISTRIBUTING MASS OVER ALL PARTICLES
*/

namespace MLE::MLPE {
	namespace rbp {

		template<typename T>
		thrust::device_vector<T> MLPE_RBP_massDistribution::copy_vec(std::vector<T> vec) {
			thrust::host_vector<T> th_vec;
			thrust::copy(vec.begin(), vec.end(), th_vec.begin());
			thrust::device_vector<T> res = th_vec;

			return res;
		}

		template<typename T>
		void MLPE_RBP_massDistribution::checkVector(std::vector<T> p) {
			thrust::device_vector<T> device_vec = copy_vec(p);
			thrust::device_vector<T>::iterator it;
			it = thrust::find_if(
				device_vec.begin(),
				device_vec.end(),
				greater_than_one());

			except::checkIfEnded(it, device_vec.end());
		}

		// needed to build a mass element - mass per particle + particle - from RigidBodyInfo - can be further optimized!
		void MLPE_RBP_massDistribution::distributeMassElements(mlpe_rbp_RigidBodyDynamicsInfo RigidBodyInfo) {
			// define transformed vectors
			thrust::device_vector<float> prob;
			std::vector<massElement> distribVec;
			// needed to transform massDistrib<float> to massDistrib2<massElement>
			thrust_wrapper_transform(
				true,
				massDistrib.prob.begin(),
				massDistrib.prob.end(),
				prob.begin(),
				multiplyByConstant<float>(RigidBodyInfo.mass));
			
			thrust_wrapper_transform(
				true,
				prob.begin(),
				prob.end(),
				RigidBodyInfo.particleDecomposition.particleDecomposition.begin(),
				RigidBodyInfo.particleDecomposition.particleDecomposition.end(),
				distribVec.begin(),
				mElementComb());
			massDistribution.massElements = distribVec;
		}

		void MLPE_RBP_massDistribution::Mass(mlpe_rbp_RigidBodyDynamicsInfo& RigidBodyInfo, float mass) {
			RigidBodyInfo.mass = mass;
		}


		void MLPE_RBP_massDistribution::getCenterMass(mlpe_rbp_RigidBodyDynamicsInfo& RigidBodyInfo) {
			glm::vec3 sum_vec = thrust_wrapper_reduce(
				true,
				massDistribution.massElements.begin(),
				massDistribution.massElements.end(),
				glm::vec3(0),
				thrust_add_Positions<massElement>());
				
			sum_vec *= 1 / RigidBodyInfo.mass;
			massDistribution.centerMass = sum_vec;
		}

		void MLPE_RBP_massDistribution::massElementsDistribution(mlpe_rbp_RigidBodyDynamicsInfo& RigidBodyInfo) {
			RigidBodyInfo.massDistribution = massDistribution;
		}

	}
}